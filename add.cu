
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add (int *a, int *b, int *c){
  *c = *a + *b;
}

int main(void){
  int a, b, c;
  int *dev_a, *dev_b, *dev_c;
  int size = sizeof(int);
  
  //  allocate device copies of a,b, c
  hipMalloc ( (void**) &dev_a, size);
  hipMalloc ( (void**) &dev_b, size);
  hipMalloc ( (void**) &dev_c, size);
  
  a=2; b=7;
  //  copy inputs to device
  hipMemcpy (dev_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy (dev_b, &b, size, hipMemcpyHostToDevice);
  
  // launch add() kernel on GPU, passing parameters
  add <<< 1, 1 >>> (dev_a, dev_b, dev_c);
  
  // copy device result back to host
  hipMemcpy (&c, dev_c, size, hipMemcpyDeviceToHost);
  std::cout<<a<<"+"<<b<<"="<<c<<std::endl;
  
  hipFree ( dev_a ); hipFree ( dev_b ); hipFree ( dev_c );
}